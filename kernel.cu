#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#include <thrust/for_each.h>
#include <thrust/execution_policy.h>

#include <stdio.h>
#include <iostream>

#include "quantities.cuh"
#include "particle.cuh"

constexpr unsigned BLK_SIZE = 256;

__global__
void interpar_compute(particle* particles, force_type* forces)
{
	__shared__ particle par_cache[BLK_SIZE];
	__shared__ force_type force_cache[BLK_SIZE];

	unsigned lc_index = threadIdx.x;
	unsigned gl_index = threadIdx.x + blockIdx.x * blockDim.x;
	par_cache[lc_index] = particles[gl_index];
	force_cache[lc_index] = forces[gl_index];

	__syncthreads();

	for (unsigned index = lc_index;
		(++index %= blockDim.x) != lc_index; )
		force_cache[lc_index] += par_cache[index].force_on(par_cache[lc_index]);

	__shared__ particle par_temp[BLK_SIZE];
	for (unsigned other_index = blockDim.x * gridDim.x + lc_index;
		other_index != lc_index; )
	{
		other_index -= blockDim.x;

		if (other_index == gl_index)
			continue;

		par_temp[lc_index] = particles[other_index];
		__syncthreads();

		unsigned index = lc_index;
		do
			force_cache[lc_index] += par_temp[index].force_on(par_cache[lc_index]);
		while ((++index %= blockDim.x) != lc_index);
	}
	__syncthreads();
	forces[gl_index] = force_cache[lc_index];

	printf("Exit %i\n", lc_index);
}

void compute(thrust::device_vector<particle>& particles,
			 thrust::device_vector<force_type>& forces)
{
	assert(particles.size() % BLK_SIZE == 0);
	assert(particles.size() == forces.size());
	interpar_compute <<< particles.size() / BLK_SIZE, BLK_SIZE >>>
		(thrust::raw_pointer_cast(particles.data()),
			thrust::raw_pointer_cast(forces.data()));
}

void advance(thrust::device_vector<particle>& particles,
			 thrust::device_vector<force_type>& forces)
{
	assert(particles.size() == forces.size());
	auto begin = thrust::make_zip_iterator(thrust::make_tuple(particles.begin(), forces.begin())),
		 end   = thrust::make_zip_iterator(thrust::make_tuple(particles.end(), forces.end()));
	thrust::for_each(thrust::device, begin, end,
		[] __device__(thrust::tuple<particle&, force_type&> tpl)
		{
			tpl.get<0>().advance(tpl.get<1>());
		});
}

int main()
{
	thrust::device_vector<particle> pts(1024 * 32);
	thrust::device_vector<force_type> forces(1024 * 32);

	std::cout << "Computation started...\n";

	compute(pts, forces);
	hipDeviceSynchronize();

	std::cout << "Advancing started...\n";

	advance(pts, forces);

	std::cout << "Ready!\n";

	/*	Output the forces:
	thrust::host_vector<force_type> h_forces = forces;

	for (force_type& force : h_forces)
		std::cout << force.force.x << ' ' << force.force.y << ' ' << force.force.z;
	*/
}