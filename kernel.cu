#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>

#include "quantities.cuh"
#include "particle.cuh"

__constant__ time_type time_step = 1.0e-4;
constexpr unsigned BLK_SIZE = 256;

__global__
void interpar_compute(particle* particles, inter_type* forces)
{
	__shared__ particle current[BLK_SIZE];
	__shared__ inter_type force_temp[BLK_SIZE];

	unsigned lc_index = threadIdx.x;
	unsigned gl_index = threadIdx.x + blockIdx.x * blockDim.x;
	current[lc_index] = particles[gl_index];
	force_temp[lc_index] = inter_type();

	for (unsigned i = lc_index; i != lc_index; )
	{
		if (++i == blockDim.x)
			i = 0;
		//TODO
	}

	__shared__ particle par_temp[BLK_SIZE];
	for (unsigned other_index = blockDim.x * (gridDim.x - 1) + lc_index;
		other_index != gl_index;
		other_index -= blockDim.x)
	{
		par_temp[lc_index] = particles[other_index];
		unsigned i = lc_index;
		do
		{
			//TODO
			if (++i == blockDim.x)
				i = 0;
		} while (i != lc_index);
	}

	//TODO
}

__inline__
void compute(thrust::device_vector<particle>& particles, thrust::device_vector<inter_type>& forces)
{
	assert(particles.size() % BLK_SIZE == 0);
	assert(particles.size() == forces.size());
	interpar_compute <<< BLK_SIZE, particles.size() / BLK_SIZE >>>
		(thrust::raw_pointer_cast(particles.data()),
		 thrust::raw_pointer_cast(forces.data()));
}

int main()
{
	//TODO
}