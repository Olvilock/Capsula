#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>

#include "quantities.cuh"
#include "particle.cuh"

constexpr unsigned BLK_SIZE = 128;

__global__
void interpar_compute(particle* particles, force_type* forces)
{
	__shared__ particle par_cache[BLK_SIZE];
	__shared__ force_type force_cache[BLK_SIZE];

	unsigned lc_index = threadIdx.x;
	unsigned gl_index = threadIdx.x + blockIdx.x * blockDim.x;
	par_cache[lc_index] = particles[gl_index];
	force_cache[lc_index] = forces[gl_index];

	__syncthreads();

	for (unsigned index = lc_index;
			   (++index %= blockDim.x) != lc_index; )
		force_cache[lc_index] += par_cache[index].force_on(par_cache[lc_index]);

	__shared__ particle par_temp[BLK_SIZE];
	for (unsigned other_index = blockDim.x * (gridDim.x - 1) + lc_index;
		other_index != lc_index;
		other_index -= blockDim.x)
	{
		if (other_index == gl_index)
			continue;

		par_temp[lc_index] = particles[other_index];
		__syncthreads();

		unsigned index = lc_index;
		do
			force_cache[lc_index] += par_temp[index].force_on(par_cache[lc_index]);
		while ((++index %= blockDim.x) != lc_index);
	}
	__syncthreads();
	forces[gl_index] = force_cache[lc_index];
}

void compute(thrust::device_vector<particle>& particles,
			 thrust::device_vector<force_type>& forces)
{
	assert(particles.size() % BLK_SIZE == 0);
	assert(particles.size() == forces.size());
	interpar_compute <<< BLK_SIZE, particles.size() / BLK_SIZE >>>
		(thrust::raw_pointer_cast(particles.data()),
		 thrust::raw_pointer_cast(forces.data()));
}

int main()
{
	thrust::device_vector<particle> pts(256);
	thrust::device_vector<force_type> forces(256);

	compute(pts, forces);
}