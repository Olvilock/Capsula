#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/execution_policy.h>

#include <stdio.h>

#include "quantities.cuh"

__constant__ time_type time_step = 1e-5;

#include "particle.cuh"

constexpr unsigned BLK_SIZE = 128;

//Call with 1-dimensional block with blockDim.x == BLK_SIZE only
__global__
void interpar_compute(particle* particles, force_type* forces, unsigned* locks)
{
	unsigned local_id = threadIdx.x;
	unsigned global_id = local_id + BLK_SIZE * blockIdx.y;
	unsigned global_cur = local_id + BLK_SIZE * blockIdx.x;

	force_type force{};
	particle par = particles[global_cur];

	__shared__ particle par_cache[BLK_SIZE];

	par_cache[local_id] = particles[global_id];
	__syncthreads();

	for (unsigned index = local_id + BLK_SIZE - (global_cur == global_id);
		index != local_id; --index)
		force += par_cache[index % BLK_SIZE].force_on(par);

	if (local_id == 0)
		while (atomicCAS(locks + blockIdx.x, gridDim.x, blockIdx.x) != blockIdx.x);
	__syncthreads();

	forces[global_cur] += force;
	__threadfence();

	if (local_id == 0)
		locks[blockIdx.x] = gridDim.x;
	__syncthreads();
}

void compute(thrust::device_vector<particle>& particles,
			 thrust::device_vector<force_type>& forces)
{
	assert(particles.size() % BLK_SIZE == 0);
	assert(particles.size() == forces.size());

	unsigned blk_dim = particles.size() / BLK_SIZE;
	thrust::device_vector<unsigned> locks(blk_dim, blk_dim);

	interpar_compute <<< dim3(blk_dim, blk_dim), BLK_SIZE >>>
		(thrust::raw_pointer_cast(particles.data()),
			thrust::raw_pointer_cast(forces.data()),
			thrust::raw_pointer_cast(locks.data()));
}

void advance(thrust::device_vector<particle>& particles,
			 thrust::device_vector<force_type>& forces)
{
	assert(particles.size() == forces.size());

	thrust::for_each_n(thrust::device,
		thrust::make_zip_iterator(thrust::make_tuple(particles.begin(), forces.begin())),
		particles.size(),
		[] __device__(thrust::tuple<particle&, force_type&> tpl)
		{
			tpl.get<0>().advance(tpl.get<1>());
			tpl.get<1>().reset();
		});
}

int main()
{
	thrust::device_vector<particle> pts(1024 * 64);
	thrust::device_vector<force_type> forces(1024 * 64);

	for (int i = 0; i < 10; ++i)
	{
		std::cout << "Computation " << i << " started...\n";
		compute(pts, forces);

		std::cout << (force_type)forces[0] << std::endl;

		advance(pts, forces);
	}
}