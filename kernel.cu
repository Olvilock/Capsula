#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>

#include "quantities.cuh"
#include "particle.cuh"

__constant__ time_type time_step = 1.0e-4;
constexpr unsigned BLK_SIZE = 256;

__global__
void interpar_compute(particle* particles, force_type* forces)
{
	__shared__ particle par_cache[BLK_SIZE];
	__shared__ force_type force_cache[BLK_SIZE];

	unsigned lc_index = threadIdx.x;
	unsigned gl_index = threadIdx.x + blockIdx.x * blockDim.x;
	par_cache[lc_index] = particles[gl_index];
	force_cache[lc_index] = forces[gl_index];

	__syncthreads();

	for (unsigned index = lc_index; index != lc_index; )
	{
		if (++index == blockDim.x)
			index = 0;

		force_type cur_force = par_cache[lc_index].force_on(par_cache[index]);
		force_cache[index] += cur_force;
		force_cache[lc_index] -= cur_force;
	}

	__shared__ particle par_temp[BLK_SIZE];
	__shared__ force_type force_temp[BLK_SIZE];
	for (unsigned other_index = blockDim.x * (gridDim.x - 1) + lc_index;
		other_index != gl_index;
		other_index -= blockDim.x)
	{
		par_temp[lc_index] = particles[other_index];
		force_temp[lc_index] = forces[other_index];

		__syncthreads();

		unsigned index = lc_index;
		do
		{
			force_type cur_force = par_cache[lc_index].force_on(par_temp[index]);
			force_temp[index] += cur_force;
			force_cache[lc_index] -= cur_force;

			if (++index == blockDim.x)
				index = 0;
		} while (index != lc_index);

		__syncthreads();

		//TODO memory transfer force_temp -> forces
	}
	//TODO memory transfer force_cache -> forces
}

__inline__
void compute(thrust::device_vector<particle>& particles,
			 thrust::device_vector<force_type>& forces)
{
	assert(particles.size() % BLK_SIZE == 0);
	assert(particles.size() == forces.size());
	interpar_compute <<< BLK_SIZE, particles.size() / BLK_SIZE >>>
		(thrust::raw_pointer_cast(particles.data()),
		 thrust::raw_pointer_cast(forces.data()));
}

int main()
{
	thrust::device_vector<particle> pts(256);
	thrust::device_vector<force_type> forces(256);

	compute(pts, forces);
}