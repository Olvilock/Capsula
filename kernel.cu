#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#include <thrust/for_each.h>
#include <thrust/execution_policy.h>

#include <stdio.h>
#include <iostream>

#include "quantities.cuh"
#include "particle.cuh"

constexpr unsigned BLK_SIZE = 128;

//Call with 1-dimensional block with blockDim.x == BLK_SIZE only
__global__
void interpar_compute(particle* particles, force_type* forces)
{
	unsigned local_id = threadIdx.x;
	unsigned global_id = local_id + BLK_SIZE * blockIdx.x;
	force_type force = forces[global_id];
	particle par = particles[global_id];

	__shared__ particle par_cache[BLK_SIZE];

	for (unsigned group_id = BLK_SIZE * gridDim.x + local_id;
		group_id != local_id; )
	{
		group_id -= BLK_SIZE;

		par_cache[local_id] = particles[group_id];
		__syncthreads();

		for (unsigned index = local_id + BLK_SIZE - (group_id == global_id);
			index != local_id; --index)
			force += par_cache[index % BLK_SIZE].force_on(par);
	}

	forces[global_id] = force;
}

void compute(thrust::device_vector<particle>& particles,
			 thrust::device_vector<force_type>& forces)
{
	assert(particles.size() % BLK_SIZE == 0);
	assert(particles.size() == forces.size());
	interpar_compute <<< particles.size() / BLK_SIZE, BLK_SIZE >>>
		(thrust::raw_pointer_cast(particles.data()),
			thrust::raw_pointer_cast(forces.data()));
}

void advance(thrust::device_vector<particle>& particles,
			 thrust::device_vector<force_type>& forces)
{
	assert(particles.size() == forces.size());
	auto begin = thrust::make_zip_iterator(thrust::make_tuple(particles.begin(), forces.begin())),
		 end   = thrust::make_zip_iterator(thrust::make_tuple(particles.end(), forces.end()));
	thrust::for_each(thrust::device, begin, end,
		[] __device__(thrust::tuple<particle&, force_type&> tpl)
		{
			tpl.get<0>().advance(tpl.get<1>());
		});
}

int main()
{
	thrust::device_vector<particle> pts(1024 * 512);
	thrust::device_vector<force_type> forces(1024 * 512);

	std::cout << "Computation started...\n";

	compute(pts, forces);
	/*
	hipDeviceSynchronize();

	std::cout << "Advancing started...\n";

	advance(pts, forces);

	std::cout << "Ready!\n";

	//	Output the forces:
	thrust::host_vector<force_type> h_forces = forces;

	for (force_type& force : h_forces)
		std::cout << force.force.x << ' ' << force.force.y << ' ' << force.force.z;
	*/
}