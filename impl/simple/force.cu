#include "hip/hip_runtime.h"
#pragma once

#include <simple/force.cuh>

namespace simple
{
	__device__ __host__
		Force::Force(const double3& d3) : m_force(d3) {}

	__device__ __host__
		Force& Force::operator += (const Force& other)
	{
		m_force.x += other.m_force.x;
		m_force.y += other.m_force.y;
		m_force.z += other.m_force.z;
		return *this;
	}

	__device__ __host__
		void Force::reset()
	{
		m_force = { 0.0, 0.0, 0.0 };
	}
}

//For dedugging reasons
std::ostream& operator <<(std::ostream& out, const simple::Force& to_out)
{
	out << to_out.m_force.x << out.fill()
		<< to_out.m_force.y << out.fill()
		<< to_out.m_force.z;
	return out;
}