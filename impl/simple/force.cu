#include "hip/hip_runtime.h"
#pragma once

#include <simple/force.cuh>

namespace simple
{
	__device__ __host__
		force_t::force_t(const double3& d3) : m_force(d3) {}

	__device__ __host__
		force_t& force_t::operator += (const force_t& other)
	{
		m_force.x += other.m_force.x;
		m_force.y += other.m_force.y;
		m_force.z += other.m_force.z;
		return *this;
	}

	__device__ __host__
		void force_t::reset()
	{
		m_force = { 0.0, 0.0, 0.0 };
	}
}

//For dedugging reasons
std::ostream& operator <<(std::ostream& out, const simple::force_t& to_out)
{
	out << to_out.m_force.x << out.fill()
		<< to_out.m_force.y << out.fill()
		<< to_out.m_force.z;
	return out;
}