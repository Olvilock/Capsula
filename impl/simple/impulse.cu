#include "hip/hip_runtime.h"
#pragma once

#include <simple/impulse.cuh>

namespace simple
{
	__device__ __host__
		Impulse::Impulse(const double3& d3) : m_impulse(d3) {}

	__device__ __host__
		Impulse& Impulse::operator += (const Impulse& other)
	{
		m_impulse.x += other.m_impulse.x;
		m_impulse.y += other.m_impulse.y;
		m_impulse.z += other.m_impulse.z;
		return *this;
	}

	__device__ __host__
		void Impulse::reset()
	{
		m_impulse = { 0.0, 0.0, 0.0 };
	}
}

//For dedugging reasons
std::ostream& operator <<(std::ostream& out, const simple::Impulse& to_out)
{
	out << to_out.m_impulse.x << out.fill()
		<< to_out.m_impulse.y << out.fill()
		<< to_out.m_impulse.z;
	return out;
}