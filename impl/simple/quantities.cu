#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_runtime.h>
#include <>

#include <simple/quantities.cuh>

namespace simple
{
	__device__ __host__
		double3 operator -(const double3& value)
	{
		return { -value.x, -value.y, -value.z };
	}

	__device__ __host__
		double3 operator -(const double3& one, const double3& other)
	{
		return { one.x - other.x, one.y - other.y, one.z - other.z };
	}

	__device__ __host__
		double3 operator *(const double& scalar, const double3& vector)
	{
		return { scalar * vector.x, scalar * vector.y, scalar * vector.z };
	}

	__device__ __host__
		double operator *(const double3& one, const double3& other)
	{
		return one.x * other.x + one.y * other.y + one.z * other.z;
	}

	__device__ __host__
		double3& operator +=(double3& value, const double3& diff)
	{
		value.x += diff.x;
		value.y += diff.y;
		value.z += diff.z;
		return value;
	}
}