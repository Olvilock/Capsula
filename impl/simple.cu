﻿//Implementation of simple

#include "simple/quantities.cuh"
#include "simple/constants.cuh"
#include "simple/force.cuh"
#include "simple/impulse.cuh"
#include "simple/wall.cuh"
#include "simple/particle.cuh"
#include "simple/advancer.cuh"

#include <simple/particle_system.cuh>

namespace simple
{
	//Explicit instantiation definition
	template struct particle_system<particle_t>;
}