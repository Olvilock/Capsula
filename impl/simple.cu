﻿//Implementation of simple

#include "general/particle_system.cuh"
#include "general/normal_pairing.cuh"

#include "simple/quantities.cuh"
#include "simple/particle.cuh"
#include "simple/advancer.cuh"

#include <simple/particle_system.cuh>

namespace simple
{
	//Explicit instantiation definition
	template struct particle_system<particle>;
}