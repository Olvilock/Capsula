﻿//Implementation of simple

#include "general/particle_system.cuh"
#include "general/normal_pairing.cuh"

#include "simple/quantities.cuh"
#include "simple/constants.cuh"
#include "simple/force.cuh"
#include "simple/impulse.cuh"
#include "simple/wall.cuh"
#include "simple/particle.cuh"
#include "simple/advancer.cuh"

#include <simple/particle_system.cuh>

namespace simple
{
	//Explicit instantiation definition
	template struct particle_system<particle_t>;
}