﻿//Implementation of simple

#include "general/particle_system.cuh"
#include "general/compute_all_forces.cuh"

#include "simple/quantities.cuh"
#include "simple/particle.cuh"
#include "simple/advancer.cuh"

#include <simple/particle_system.cuh>

namespace simple
{
	//Specify template for code generation;
	template struct particle_system<particle>;
}