﻿//Implementation of simple

#include "general/particle_system.cuh"
#include "general/normal_pairing.cuh"

#include "simple/quantities.cuh"
#include "simple/constants.cuh"
#include "simple/force.cuh"
#include "simple/wallfrc.cuh"
#include "simple/particle.cuh"
#include "simple/wall.cuh"
#include "simple/advancer.cuh"

#include <simple/particle_system.cuh>

namespace simple
{
	//Explicit instantiation definition
	template struct particle_system<particle_t>;
}