//Implementation of particle_system

#include <hip/hip_runtime.h>
#include <>

#include <particle_system.cuh>
#include <LIBS.cuh>

template</*particle*/ typename particle_t, /*advancer*/ typename advancer_t>
particle_system<particle_t, advancer_t>::particle_system(size_t size) :
	m_particles(size),
	m_forces(size) {}

template</*particle*/ typename particle_t, /*advancer*/ typename advancer_t>
void particle_system<particle_t, advancer_t>::compute()
{
	assert(m_particles.size() % BLK_SIZE == 0);
	assert(m_particles.size() == m_forces.size());

	unsigned blk_dim = m_particles.size() / BLK_SIZE;
	thrust::device_vector<unsigned> locks(blk_dim, blk_dim);

	std::lock_guard<std::mutex> lock_system(m_mutex);

	compute_interparticle_forces<particle_t> <<< dim3(blk_dim, blk_dim), BLK_SIZE >>>
		(thrust::raw_pointer_cast(m_particles.data()),
			thrust::raw_pointer_cast(m_forces.data()),
			thrust::raw_pointer_cast(locks.data()));
}

template</*particle*/ typename particle_t, /*advancer*/ typename advancer_t>
void particle_system<particle_t, advancer_t>::advance()
{
	assert(m_particles.size() == m_forces.size());

	std::lock_guard<std::mutex> lock_system(m_mutex);

	thrust::transform(thrust::device,
		m_particles.cbegin(),
		m_particles.cend(),
		m_forces.begin(),
		m_particles.begin(),
		m_advancer);
}
