﻿//Implementation of kernel functions

#include "quantities.cuh"
#include "kernels/quantities.cuh"

#include "particle.cuh"
#include "kernels/particle.cuh"

#include "particle_advancer.cuh"
#include "kernels/particle_advancer.cuh"

#include "particle_system.cuh"
#include "kernels/particle_system.cuh"
