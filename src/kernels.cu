﻿//Implementation of kernel functions

#include <quantities.cuh>
#include <kernels/quantities.cuh>

#include <particle.cuh>
#include <kernels/particle.cuh>

#include <particle_advancer.cuh>
#include <kernels/particle_advancer.cuh>

#include <compute_forces.cuh>
#include <kernels/compute_forces.cuh>
