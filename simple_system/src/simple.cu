﻿//Implementation of simple

#include <default/impl/particle_system.cuh>
#include <default/impl/compute_all_forces.cuh>

#include "impl/quantities.cuh"
#include "impl/particle.cuh"
#include "impl/advancer.cuh"

#include <simple/particle_system.cuh>

namespace simple
{
	//Specify template for code generation;
	template struct particle_system<particle, advancer>;
}